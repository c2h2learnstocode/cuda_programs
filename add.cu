
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>



void init(float *x, int s){
    int i=0;
    for(i=0; i<s; i++){
        x[i]=1.0f * (float)i;
    }
}


__global__
void compute(float *x, float *y, int s){
    int i=0;
    for(i=0; i<s; i++){
      y[i]= x[i]*x[i]; 
    }
}

int main(){
    int N = 1<<20;
    
    float *x;// = malloc(sizeof(float)*N);
    float *y;// = malloc(sizeof(float)*N);
    
    hipMallocManaged(&x, sizeof(float)*N);
    hipMallocManaged(&y, sizeof(float)*N);

    init(x, N);
    init(y, N);
    int i=0;
    compute<<<1,1>>>(x, y, N);

    hipDeviceSynchronize();

    for(i=0; i<N; i++){
        printf("%d %f %f\n", i, x[i], y[i]);

    }
}
